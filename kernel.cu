﻿// Artem Fomin

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <stdlib.h>
#include <time.h>


struct Vector {
	int n;
	int* data;
};

struct Vector* new_vector_cpu(int n, bool random = false) {
	struct Vector* vector = (struct Vector*)malloc(sizeof(struct Vector));

	vector->n = n;
	vector->data = (int*)malloc(n * sizeof(int));

	srand(time(NULL));
	for (int i = 0; i < n; i++) {
		if (random) {
			vector->data[i] = rand() % 100;
		} else {
			vector->data[i] = 0;
		}
	}

	return vector;
}

struct Vector* new_vector_gpu(int n) {
	struct Vector* vector = nullptr;

	hipMalloc((void**)&vector, sizeof(struct Vector));
	hipMalloc((void**)(&(vector->data)), n * sizeof(int));

	hipMemcpy((void*)(vector->n), &n, sizeof(int), hipMemcpyHostToDevice);

	int* data = (int*)malloc(n * sizeof(int));
	for (int i = 0; i < n; i++) {
		data[i] = 0;
	}
	hipMemcpy((void*)(vector->data), data, n * sizeof(int), hipMemcpyHostToDevice);

	free(data);

	return vector;
}

void free_vector_cpu(struct Vector* vector) {
	if (vector->data != nullptr) {
		free(vector->data);
	}

	if (vector != nullptr) {
		free(vector);
	}
}

void free_vector_gpu(struct Vector* vector) {
	if (vector->data != nullptr) {
		hipFree(vector->data);
	}

	if (vector != nullptr) {
		hipFree(vector);
	}
}

int add_vectors_on_cpu(struct Vector* const a, struct Vector* const b, struct Vector* c) {
	if (a->n != b->n) {
		return 1;
	}

	int n = a->n;
	c = new_vector_cpu(n);
	for (int i = 0; i < n; i++) {
		c->data[i] = a->data[i] + b->data[i];
	}

	return 0;
}

void copy_vector_h_to_d(struct Vector* const vec_h, struct Vector** vec_d) {
	*vec_d = new_vector_gpu(vec_h->n);

	hipMemcpy((void*)((*vec_d)->n), &(vec_h->n), sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy((void*)((*vec_d)->data), &(vec_h->data), vec_h->n * sizeof(int), hipMemcpyHostToDevice);
}



int add_vectors_on_gpu(struct Vector* a_h, struct Vector* b_h, struct Vector* c_h) {
	if (a_h->n != b_h->n) {
		return 1;
	}

	int n = a_h->n;
	struct Vector *a_d, *b_d, *c_d;

	copy_vector_h_to_d(a_h, &a_d);
	copy_vector_h_to_d(b_h, &b_d);
	c_d = new_vector_gpu(n);
}

__global__ void add_vectors_d(struct Vector* a, struct Vector* b, struct Vector* c) {
	c->data[threadIdx.x] = a->data[threadIdx.x] + b->data[threadIdx.x];
}

int main(void) {

	return 0;
}
